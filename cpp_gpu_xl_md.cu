
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>

static const unsigned n_particles = 2000;

__global__
void evaluate(double *x, double *y, double *z, unsigned *ppairs,
              double *scores, unsigned n_ppairs) {
  static const double mean = 2.0;
  static const double force = 0.1;
  unsigned start = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned stride = blockDim.x * gridDim.x;
  for (unsigned i = start; i < n_ppairs; i += stride) {
    double dx = x[ppairs[i*2]] - x[ppairs[i*2+1]];
    double dy = y[ppairs[i*2]] - y[ppairs[i*2+1]];
    double dz = z[ppairs[i*2]] - z[ppairs[i*2+1]];
    double r = sqrt((dx*dx) + (dy*dy) + (dz*dz));
    scores[i] = 0.5 * force * (r - mean) * (r - mean);
  }
}

__global__
void integrate(double *x, unsigned n_particles) {
  unsigned start = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned stride = blockDim.x * gridDim.x;
  for (unsigned p = start; p < n_particles; p += stride) {
    if (p % 2 == 0) {
      x[p] += 0.01;
    } else {
      x[p] -= 0.01;
    }
  }
}

double optimize(unsigned max_steps, double *x, double *y, double *z,
                unsigned *ppairs, double *scores, unsigned n_ppairs) {
  double score = 0.;
  int block_size = 256;
  int num_blocks_eval = (n_ppairs + block_size - 1) / block_size;
  int num_blocks_opt = (n_particles + block_size - 1) / block_size;
  for (unsigned step = 0; step < max_steps; ++step) {
    /*score = */evaluate<<<num_blocks_eval, block_size>>>(x, y, z, ppairs, scores, n_ppairs);
    // cudaDeviceSynchronize(); ?
    integrate<<<num_blocks_opt, block_size>>>(x, n_particles);
    // cudaDeviceSynchronize(); ?
  }
  hipDeviceSynchronize();
  return score;
}

int main() {
  double *x, *y, *z, *scores;
  unsigned *ppairs;
  hipMallocManaged(&x, n_particles * sizeof(double));
  hipMallocManaged(&y, n_particles * sizeof(double));
  hipMallocManaged(&z, n_particles * sizeof(double));
  unsigned n_ppairs = n_particles - 1;
  hipMallocManaged(&ppairs, n_ppairs * 2 * sizeof(unsigned));
  hipMallocManaged(&scores, n_ppairs * sizeof(double));

  for (unsigned i = 0; i < n_particles; ++i) {
    x[i] = 0.01 * i;
    y[i] = 0.0;
    z[i] = 0.0;
  }

  unsigned k = 0;
  for (unsigned i = 0; i < n_particles - 1; ++i) {
    ppairs[k++] = i;
    ppairs[k++] = i + 1;
  }

  int block_size = 256;
  int num_blocks = (n_ppairs + block_size - 1) / block_size;

  evaluate<<<num_blocks, block_size>>>(x, y, z, ppairs, scores, n_ppairs);
  hipDeviceSynchronize();
  double score = 0.;
  for (unsigned i = 0; i < n_ppairs; ++i) {
    score += scores[i];
  }
  std::cerr << "Initial score " << score << std::endl;

  auto timestart = std::chrono::steady_clock::now();
  optimize(1000000, x, y, z, ppairs, scores, n_ppairs);
  auto timeend = std::chrono::steady_clock::now();
  auto span = std::chrono::duration_cast<
                      std::chrono::duration<double> >(timeend - timestart);

  std::cerr << "Optimization time: " << span.count() << std::endl;

    hipDeviceSynchronize();
  evaluate<<<num_blocks, block_size>>>(x, y, z, ppairs, scores, n_ppairs);
  hipDeviceSynchronize();
  score = 0.;
  for (unsigned i = 0; i < n_ppairs; ++i) {
    score += scores[i];
  }
  std::cerr << "Final score " << score << std::endl;

  return 0;
}
